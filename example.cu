#include "hip/hip_runtime.h"
// compile with `nvcc --std=c++14 example.cu`

#include <cstdio>
#include "default_memory_resource.hpp"

constexpr int size{10};

__global__ void init(int* data) {
  if (threadIdx.x < size) {
    data[threadIdx.x] = threadIdx.x;
  }
}

__global__ void print(int* data) {
  if (threadIdx.x < size) {
    printf("%d\n", data[threadIdx.x]);
  }
}

int main(void) {
  rmm::mr::device_memory_resource* resource = rmm::mr::get_default_resource();

  void* data = resource->allocate(size * sizeof(int));

  init<<<1, 256>>>(static_cast<int*>(data));
  print<<<1, 256>>>(static_cast<int*>(data));

  resource->deallocate(data, size*sizeof(int));
}